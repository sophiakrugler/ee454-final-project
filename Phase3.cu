#include "hip/hip_runtime.h"
// This is Phase 3 of the EE454 Final Project

// The goal is to build a convolutional layer and compare it's results

// Define the convolution as a kernel which can be done in parallel
#define STARTING_SIZE 28
#define WINDOW_SIZE 3

__global__ void Convolution(int i_featuremap[STARTING_SIZE][STARTING_SIZE], int o_featuremap[STARTING_SIZE-WINDOW_SIZE+1][STARTING_SIZE-WINDOW_SIZE+1],  int kernel[WINDOW_SIZE][WINDOW_SIZE])
{
    // This will take in a featuremap, apply a filter (kernel), and output an element of the resulting featuremap
    // The input featuremap is STARTING_SIZE x STARTING SIZE, or 28x28, the output is (STARTING_SIZE - WINDOW_SIZE + 1) x (STARTING_SIZE - WINDOW_SIZE + 1), or 26x26

    // Extract the 3x3 window from i_featuremap

    // Multiply Window Element-wise by the Kernel

    // Sum the resulting array

    // Return that sum as an element of o_featuremap
}

int main()
{
    // TODO: determine how many convolutions are done in parallel
    int total_threads = (STARTING_SIZE-WINDOW_SIZE+1)*(STARTING_SIZE-WINDOW_SIZE+1);

    // invoke the kernel for a number of threads
    Convolution<<<1 , total_threads>>>(i_featuremap, kernel);
    return 0;
}