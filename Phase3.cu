#include "hip/hip_runtime.h"
﻿
// This is Phase 3 of the EE454 Final Project

// The goal is to build a convolutional layer and compare it's results

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// Define the convolution as a kernel which can be done in parallel
#define STARTING_SIZE  28
#define ENDING_SIZE    26
#define WINDOW_SIZE    3

hipError_t convWithCuda(int* i_map, int* kernel, int* o_map);
void create_input_featuremap(int* featuremap);
void create_kernel(int* kernel);

//__global__ void Convolution(int i_featuremap[STARTING_SIZE][STARTING_SIZE], int o_featuremap[STARTING_SIZE - WINDOW_SIZE + 1][STARTING_SIZE - WINDOW_SIZE + 1], int kernel[WINDOW_SIZE][WINDOW_SIZE])
__global__ void Convolution(int* i_featuremap, int* o_featuremap, int* kernel)
{
    // This will take in a featuremap, apply a filter (kernel), and output an element of the resulting featuremap
    // The input featuremap is STARTING_SIZE x STARTING SIZE, or 28x28, the output is (STARTING_SIZE - WINDOW_SIZE + 1) x (STARTING_SIZE - WINDOW_SIZE + 1), or 26x26

    // Extract the 3x3 window from i_featuremap
    int output_element = threadIdx.x; // which element we are currently computing
    // element i corresponds to the window: from i_row  to i_row + WINDOW_SIZE - 1 by i_column to i_column + WINDOW_SIZE - 1
    int sum = 0;
    int out_row = output_element / ENDING_SIZE; // row and column of this element in the output featuremap
    int out_column = output_element % ENDING_SIZE;


    for (int i = 0; i < WINDOW_SIZE; i++)
    {
        for (int j = 0; j < WINDOW_SIZE; j++)
        {
            // Multiply Window Element-wise by the Kernel and Sum the result
            sum = sum + i_featuremap[(out_row + i) * STARTING_SIZE + (out_column + j)] * kernel[i * WINDOW_SIZE + j];
        }
    }
    o_featuremap[output_element] = sum;
}

int main()
{
    // Make i_featuremap
    int i_featuremap[STARTING_SIZE * STARTING_SIZE] = { 0 }; // TODO: Need to initialize this to something
    create_input_featuremap(i_featuremap);
    // Print the input map
    for (int i = 0; i < STARTING_SIZE; i++)
    {
        for (int j = 0; j < STARTING_SIZE; j++)
        {
            printf("{%d}", i_featuremap[(i * STARTING_SIZE) + j]);
        }
        printf("\n");
    }
    printf("\n");

    // Initialize o_featuremap
    int o_featuremap[(STARTING_SIZE - WINDOW_SIZE + 1) * (STARTING_SIZE - WINDOW_SIZE + 1)] = { 0 }; // TODO: Need to initialize this to something

    // Make kernel
    int kernel[WINDOW_SIZE * WINDOW_SIZE] = { 0 }; // Need to initialize this to something
    create_kernel(kernel);
    // Print the kernel
    for (int i = 0; i < WINDOW_SIZE; i++)
    {
        for (int j = 0; j < WINDOW_SIZE; j++)
        {
            printf("{%d}", kernel[(i * WINDOW_SIZE) + j]);
        }
        printf("\n");
    }
    printf("\n");

    // Apply convolutions in parallel
    hipError_t cudaStatus = convWithCuda(i_featuremap, kernel, o_featuremap);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // Print the output map
    for (int i = 0; i < ENDING_SIZE; i++)
    {
        for (int j = 0; j < ENDING_SIZE; j++)
        {
            printf("{%d}", o_featuremap[(i * ENDING_SIZE) + j]);
        }
        printf("\n");
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

void create_input_featuremap(int* featuremap)
{
    for (int i = 0; i < STARTING_SIZE; i++)
    {
        for (int j = 0; j < STARTING_SIZE; j++)
        {
            featuremap[(i * STARTING_SIZE) + j] = j % (STARTING_SIZE / 2); // Creates vertical lines
        }
    }
}

void create_kernel(int* kernel)
{
    for (int i = 0; i < WINDOW_SIZE; i++)
    {
        for (int j = 0; j < WINDOW_SIZE; j++)
        {
            if (i != j)
            {
                kernel[(i * WINDOW_SIZE) + j] = 2;
            }
            else
            {
                kernel[(i * WINDOW_SIZE) + j] = 0;
            }
        }
    }
}

hipError_t convWithCuda(int* i_map, int* kernel, int* o_map)
{
    int* dev_input = 0;
    int* dev_output = 0;
    int* dev_kernel = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for the 3 arrays
    cudaStatus = hipMalloc((void**)&dev_output, ENDING_SIZE * ENDING_SIZE * sizeof(int)); // allocate ENDING_SIZE^2 * size of int for the output featuremap
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_input, STARTING_SIZE * STARTING_SIZE * sizeof(int)); // allocate STARTING_SIZE^2 * size of int for the input featuremap
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_kernel, WINDOW_SIZE * WINDOW_SIZE * sizeof(int)); // allocate WINDOW_SIZE^2 * size of int for the input kernel
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy the vectors from Host to GPU buffers
    cudaStatus = hipMemcpy(dev_input, i_map, STARTING_SIZE * STARTING_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_kernel, kernel, WINDOW_SIZE * WINDOW_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element
    Convolution << <1, ENDING_SIZE*ENDING_SIZE >> > (dev_input, dev_output, dev_kernel);

    // Check for any errors when launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy the output vector from GPU buffer back to host memory
    cudaStatus = hipMemcpy(o_map, dev_output, ENDING_SIZE * ENDING_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    // Free the memory for the 3 arrays
    hipFree(dev_input);
    hipFree(dev_output);
    hipFree(dev_kernel);

    // Return whether it failed
    return cudaStatus;
}