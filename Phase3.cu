#include "hip/hip_runtime.h"
﻿
// This is Phase 3 of the EE454 Final Project

// The goal is to build a convolutional layer and compare it's results

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// Define the convolution as a kernel which can be done in parallel
#define STARTING_SIZE 5
#define ENDING_SIZE   3
#define WINDOW_SIZE    3

hipError_t convWithCuda(const int(*i_map)[STARTING_SIZE], const int(*kernel)[WINDOW_SIZE], int (*o_map)[ENDING_SIZE]);
void create_input_featuremap(int (*featuremap)[STARTING_SIZE]);
void create_kernel(int (*kernel)[WINDOW_SIZE]);

//__global__ void Convolution(int i_featuremap[STARTING_SIZE][STARTING_SIZE], int o_featuremap[STARTING_SIZE - WINDOW_SIZE + 1][STARTING_SIZE - WINDOW_SIZE + 1], int kernel[WINDOW_SIZE][WINDOW_SIZE])
__global__ void Convolution(int (*i_featuremap)[STARTING_SIZE], int (*o_featuremap)[ENDING_SIZE], int (*kernel)[WINDOW_SIZE])
{
    // This will take in a featuremap, apply a filter (kernel), and output an element of the resulting featuremap
    // The input featuremap is STARTING_SIZE x STARTING SIZE, or 28x28, the output is (STARTING_SIZE - WINDOW_SIZE + 1) x (STARTING_SIZE - WINDOW_SIZE + 1), or 26x26

    // Extract the 3x3 window from i_featuremap
    int i = threadIdx.x; // which element we are currently computing
    int sum = 0;

    for (int j = 0; j < (WINDOW_SIZE * WINDOW_SIZE); j++)
    {
        // Multiply Window Element-wise by the Kernel and Sum the result
        sum = sum + i_featuremap[(i / ENDING_SIZE) + (j / WINDOW_SIZE)][(j % ENDING_SIZE) + (j % WINDOW_SIZE)] * kernel[j / WINDOW_SIZE][j % WINDOW_SIZE];
    }

    // Return that sum as an element of o_featuremap
    o_featuremap[i / ENDING_SIZE][i % ENDING_SIZE] = sum;
}

int main()
{
    // Make i_featuremap
    int i_featuremap[STARTING_SIZE][STARTING_SIZE] = { 0 }; // TODO: Need to initialize this to something
    create_input_featuremap(i_featuremap);

    // Initialize o_featuremap
    int o_featuremap[(STARTING_SIZE - WINDOW_SIZE + 1)][(STARTING_SIZE - WINDOW_SIZE + 1)] = {0}; // TODO: Need to initialize this to something

    // Make kernel
    int kernel[WINDOW_SIZE][WINDOW_SIZE] = { 1 }; // Need to initialize this to something
    create_kernel(kernel);

    int total_threads = (STARTING_SIZE - WINDOW_SIZE + 1) * (STARTING_SIZE - WINDOW_SIZE + 1);

    hipError_t cudaStatus = convWithCuda(i_featuremap, kernel, o_featuremap);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // invoke the kernel for a number of threads
    //Convolution<<<1,total_threads>>>(i_featuremap, o_featuremap, kernel);

    // Print the input map
    for (int i = 0; i < STARTING_SIZE; i++)
    {
        for (int j = 0; j < STARTING_SIZE; j++)
        {
            printf("{%d}", i_featuremap[i][j]);
        }
        printf("\n");
    }
    printf("\n");

    // Print the kernel
    for (int i = 0; i < WINDOW_SIZE; i++)
    {
        for (int j = 0; j < WINDOW_SIZE; j++)
        {
            printf("{%d}", kernel[i][j]);
        }
        printf("\n");
    }
    printf("\n");

    // Print the output map
    for (int i = 0; i < ENDING_SIZE; i++)
    {
        for (int j = 0; j < ENDING_SIZE; j++)
        {
            printf("{%d}", o_featuremap[i][j]);
        }
        printf("\n");
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

void create_input_featuremap(int (*featuremap)[STARTING_SIZE])
{
    for (int i = 0; i < STARTING_SIZE; i++)
    {
        for (int j = 0; j < STARTING_SIZE; j++)
        {
            featuremap[i][j] = j%(STARTING_SIZE/2); // Creates vertical lines
        }
    }
}

void create_kernel(int (*kernel)[WINDOW_SIZE])
{
    for (int i = 0; i < WINDOW_SIZE; i++)
    {
        for (int j = 0; j < WINDOW_SIZE; j++)
        {
            if (i != j)
            {
                kernel[i][j] = 1;
            }
            else
            {
                kernel[i][j] = 0;
            }
        }
    }
}

hipError_t convWithCuda(const int(*i_map)[STARTING_SIZE], const int(*kernel)[WINDOW_SIZE], int (*o_map)[ENDING_SIZE])
{
    int (*dev_input)[STARTING_SIZE] = { 0 };
    int (*dev_output)[ENDING_SIZE] = { 0 };
    int (*dev_kernel)[WINDOW_SIZE] = { 0 };
    hipError_t cudaStatus;
    
    // Choose which GPU to run on
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for the 3 arrays
    cudaStatus = hipMalloc((void**)&dev_output, ENDING_SIZE * ENDING_SIZE * sizeof(int)); // allocate ENDING_SIZE^2 * size of int for the output featuremap
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_input, STARTING_SIZE * STARTING_SIZE * sizeof(int)); // allocate STARTING_SIZE^2 * size of int for the input featuremap
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_kernel, WINDOW_SIZE * WINDOW_SIZE * sizeof(int)); // allocate WINDOW_SIZE^2 * size of int for the input kernel
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy the vectors from Host to GPU buffers
    cudaStatus = hipMemcpy(dev_input, i_map, STARTING_SIZE * STARTING_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_output, o_map, ENDING_SIZE * ENDING_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_kernel, kernel, WINDOW_SIZE * WINDOW_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element
    Convolution << <1, ENDING_SIZE*ENDING_SIZE >> > (dev_input, dev_output, dev_kernel);

    // Check for any errors when launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

Error:
    // Free the memory for the 3 arrays
    hipFree(dev_input);
    hipFree(dev_output);
    hipFree(dev_kernel);

    // Return whether it failed
    return cudaStatus;
}